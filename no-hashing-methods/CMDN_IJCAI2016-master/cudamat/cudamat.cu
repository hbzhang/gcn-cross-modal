#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <math.h>
#include "cudamat_kernels.cuh"
#include "cudamat.cuh"

extern "C" {

/* ------------------------------ CUBLAS init/shutdown ------------------------------ */

inline bool check_cublas_error() {
    hipblasStatus_t status = cublasGetError();
    return status != HIPBLAS_STATUS_SUCCESS;
}

inline bool checkCUDAError() {
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
        printf("%s\n", hipGetErrorString( err));
    return hipSuccess != err;
}

extern const char* get_last_cuda_error() {
    hipError_t err = hipGetLastError();

    return hipGetErrorString( err);
}

extern int cublas_init() {
    cublasInit();
    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}

extern int cublas_shutdown() {
    cublasShutdown();
    hipDeviceReset();
    return 0;
}


extern int cuda_set_device(int deviceId) {
    hipSetDevice(deviceId);
    
    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int init_random(rnd_struct* rnd_state, int seed, char* cudamatpath) {
    unsigned int * host_mults;
    host_mults = (unsigned int*)malloc(NUM_RND_STREAMS * sizeof(unsigned int));
    FILE * pFile;

    pFile = fopen (cudamatpath,"r");

    for (int i = 0; i < NUM_RND_STREAMS; i++) {
        fscanf (pFile, "%u", &host_mults[i]);
    }
    fclose (pFile);

    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned int), (void**)&rnd_state->dev_mults);
    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned long long), (void**)&rnd_state->dev_words);
    hipblasSetVector(NUM_RND_STREAMS, sizeof(unsigned int), host_mults, 1, rnd_state->dev_mults, 1);
    //hipMalloc((void **)&rnd_state->dev_mults, NUM_RND_STREAMS * sizeof(unsigned int));
    //hipMalloc((void **)&rnd_state->dev_words, NUM_RND_STREAMS * sizeof(unsigned long long));
    //hipMemcpy(rnd_state->dev_mults, host_mults, NUM_RND_STREAMS * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    kSeedRandom<<<NUM_RND_BLOCKS, NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, seed);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

/* ------------------------------ Utility routines ------------------------------ */

extern int get_leading_dimension(cudamat* mat) {
    return mat->is_trans ? mat->size[1] : mat->size[0];
}

extern int get_nonleading_dimension(cudamat* mat) {
    return mat->is_trans ? mat->size[0] : mat->size[1];
}

extern void set_transpose(cudamat* mat, int is_trans) {
    mat->is_trans = is_trans;
}

inline char get_transpose_char(cudamat* mat) {
    return mat->is_trans ? 't' : 'n';
}

extern void cuda_sync_threads() {
    hipDeviceSynchronize();
}

/* ------------------------------ Allocating/moving data ------------------------------ */

extern int allocate_device_memory(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];

    hipblasStatus_t stat;

    stat = cublasAlloc(len, sizeof(mat->data_device[0]), (void**)&mat->data_device);

    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    mat->on_device = 1;
    return 0;
}

extern int allocate_device_memory_sparse(cudamat_sparse* mat) {
    int nnz = mat->nnz, rows = mat->size[0];

    hipblasStatus_t stat;

    stat = cublasAlloc(nnz, sizeof(mat->data_device.data[0]), (void**)&mat->data_device.data);
    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    stat = cublasAlloc(nnz, sizeof(mat->data_device.indices[0]), (void**)&mat->data_device.indices);
    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    stat = cublasAlloc(rows + 1, sizeof(mat->data_device.indptr[0]), (void**)&mat->data_device.indptr);
    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    mat->on_device = 1;
    return 0;
}


extern int copy_to_host(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];

    if (mat->on_device) {
            hipblasGetVector(len, sizeof(mat->data_host[0]), mat->data_device, 1, mat->data_host, 1);

        if (check_cublas_error())
            return CUBLAS_ERROR;
    } else
       return ERROR_NOT_ON_DEVICE;
 
    return 0;
}

extern int copy_to_device(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];
    int err_code = 0;

    //if (!mat->owns_data)
    //    return VIEW_ERROR;

    if (!mat->on_device) {
        err_code = allocate_device_memory(mat);
        if (err_code)
            return err_code;
    }

    hipblasSetVector(len, sizeof(mat->data_host[0]), mat->data_host, 1, mat->data_device, 1);
    
    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

extern int copy_sparse_to_device(cudamat_sparse* mat) {
    int len = mat->nnz, rows = mat->size[0];
    int err_code = 0;

    //if (!mat->owns_data)
    //    return VIEW_ERROR;

    if (!mat->on_device) {
        err_code = allocate_device_memory_sparse(mat);
        if (err_code)
            return err_code;
    }

    hipblasSetVector(len, sizeof(mat->data_host.data[0]), mat->data_host.data, 1, mat->data_device.data, 1);
    if (check_cublas_error())
        return CUBLAS_ERROR;

    hipblasSetVector(len, sizeof(mat->data_host.indices[0]), mat->data_host.indices, 1, mat->data_device.indices, 1);
    if (check_cublas_error())
        return CUBLAS_ERROR;

    hipblasSetVector(rows + 1, sizeof(mat->data_host.indptr[0]), mat->data_host.indptr, 1, mat->data_device.indptr, 1);
    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}


extern int copy_on_device(cudamat* mat1, cudamat* mat2) {
    int len = mat1->size[0]*mat1->size[1];

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    hipblasScopy(len, mat1->data_device, 1, mat2->data_device, 1);

    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}

extern int get_row_slice(cudamat* source, cudamat* target, unsigned int start, unsigned int end) {
    int height = source->size[0];
    int width = source->size[1];

    if ((end - start) != target->size[0] || source->size[1] != target->size[1] || start >= end || end > height)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    dim3 kernelBlockGrid((int)ceil((end - start)/32.), (int)ceil(width/32.), 1);
    dim3 kernelBlockDim(32, 1, 1);

    kGetRowSlice<<<kernelBlockGrid,kernelBlockDim>>>(source->data_device, target->data_device, start, end, width, height);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int set_row_slice(cudamat* source, cudamat* target, unsigned int start, unsigned int end) {
    int height = target->size[0];
    int width = target->size[1];

    if ((end - start) != source->size[0] || source->size[1] != target->size[1] || start >= end || end > height)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    dim3 kernelBlockGrid((int)ceil((end - start)/32.), (int)ceil(width/32.), 1);
    dim3 kernelBlockDim(32, 1, 1);

    kSetRowSlice<<<kernelBlockGrid,kernelBlockDim>>>(source->data_device, target->data_device, start, end, width, height);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int copy_transpose(cudamat* source, cudamat* target) {
    unsigned int height = source->size[0];
    unsigned int width = source->size[1];

    if (source->size[0] != target->size[1] || source->size[1] != target->size[0])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    // setup execution parameters
    unsigned int grid_x = height / COPY_BLOCK_SIZE;
    if (height % COPY_BLOCK_SIZE)
        grid_x++;

    unsigned int grid_y = width / COPY_BLOCK_SIZE;
    if (width % COPY_BLOCK_SIZE)
        grid_y++;

    dim3 grid(grid_x, grid_y, 1);
    dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);

    kTranspose<<< grid, threads >>>(target->data_device, source->data_device, height, width);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int free_device_memory(cudamat* mat) {
    if (mat->owns_data && mat->on_device) {
        hipblasStatus_t stat;

        stat = cublasFree(mat->data_device);
        mat->on_device = 0;

        if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error())
            return CUBLAS_ERROR;
    }

    return 0;
}

extern int set_shape(cudamat* mat, unsigned int m, unsigned int n) {

    mat->size[0] = m;
    mat->size[1] = n;

    return 0;
}


extern int reshape(cudamat* mat, unsigned int m, unsigned int n) {
    if (mat->size[0] * mat->size[1] != m * n)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    mat->size[0] = m;
    mat->size[1] = n;

    return 0;
}

extern int get_slice(cudamat* source, cudamat* target, unsigned int first_col, unsigned int last_col) {
    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (!source->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (last_col > source->size[1] || (first_col >= last_col))
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int num_rows = source->size[0];

    target->data_host = 0;
    target->data_device = source->data_device + first_col * num_rows;
    target->on_device = 1;
    target->on_host = 0;
    target->size[0] = source->size[0];
    target->size[1] = last_col - first_col;
    target->is_trans = 0;
    target->owns_data = 0;

    return 0;
}

extern int get_vector_slice(cudamat* source, cudamat* target, unsigned int first_ind, unsigned int last_ind) {
    // source must be a vector.
    if (source->size[0] > 1 && source->size[1] > 1)
        return ERROR_GENERIC;

    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (!source->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (first_ind >= last_ind)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int num_rows = source->size[0];

    target->data_host = 0;
    target->data_device = source->data_device + first_ind * num_rows;
    target->on_device = 1;
    target->on_host = 0;
    target->is_trans = 0;
    target->owns_data = 0;

    if (source->size[0] > 1) {
        if (last_ind > source->size[0])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        target->size[0] = last_ind - first_ind;
        target->size[1] = 1;
    } else {
        if (last_ind > source->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        target->size[0] = 1;
        target->size[1] = last_ind - first_ind;
    }

    return 0;
}

/* ------------------------------ Initialization routines ------------------------------ */

extern void init_from_array(cudamat* mat, float* data, int m, int n) {
    mat->data_host = data;
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 1;
    mat->is_trans = 0;
    mat->owns_data = 1;
}

extern void init_from_sparse_array(cudamat_sparse* mat, float* data, int* indices, int* indptr, int m, int n, int nnz) {
    mat->data_host.data = data;
    mat->data_host.indices = indices;
    mat->data_host.indptr = indptr;
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 1;
    mat->is_trans = 0;
    mat->owns_data = 1;
    mat->nnz = nnz;
}


extern void set_on_device(cudamat* mat) {
  mat->on_device = 1;
}

extern int init_empty(cudamat* mat, int m, int n) {
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 0;
    mat->is_trans = 0;
    mat->owns_data = 1;

    return allocate_device_memory(mat);
}

/* ------------------------------ Random number generation ------------------------------ */
extern int fill_with_rand(rnd_struct* rnd_state, cudamat* mat) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kRandomUniform<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int fill_with_randn(rnd_struct* rnd_state, cudamat* mat) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kRandomGaussian<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int sample_bernoulli(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kSampleBernoulli<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}
extern int sample_bernoulli_tanh(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kSampleBernoulliTanh<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}
extern int sample_poisson(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kSamplePoisson<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}
extern int sample_gaussian(rnd_struct* rnd_state, cudamat* mat, cudamat* target, float mult) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kSampleGaussian<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len, mult);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int perturb_energy(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kPerturbEnergy<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int perturb_prob(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kPerturbProb<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int dropout(rnd_struct* rnd_state, cudamat* mat, float dropprob, float val) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kRandomDropout<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len, dropprob, val);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

/* ------------------------------ Algebraic operations ------------------------------ */

extern int add_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddColVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError()) {
        return CUDA_ERROR;
    }

    return 0;
}

extern int add_col_mult(cudamat* mat, cudamat* vec, cudamat* target, float mult) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddColMult<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, mult, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_diagonal_scalar(cudamat* mat, float val, cudamat* target) {
    unsigned int w = mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultDiagonalScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, w);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


extern int add_diagonal_scalar(cudamat* mat, float val, cudamat* target) {
    unsigned int w = mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddDiagonalScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, w);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


extern int mult_diagonal(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[1] * vec->size[0] ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultDiagonal<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


extern int add_diagonal(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[1] * vec->size[0] ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddDiagonal<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


extern int add_row_mult(cudamat* mat, cudamat* vec, cudamat* target, float mult) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddRowMult<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, mult, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int add_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddRowVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_by_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultByColVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_by_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultByRowVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int div_by_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivByColVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int div_by_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivByRowVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int less_than_eq(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThanEq<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int less_than(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThan<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int less_than_eq_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThanEqScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


extern int less_than_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThanScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int greater_than_eq(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThanEq<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int greater_than(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThan<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int upper_bound(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kUpperBound<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


extern int lower_bound(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLowerBound<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int greater_than_eq_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThanEqScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int greater_than_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThanScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int upper_bound_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kUpperBoundScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


extern int lower_bound_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLowerBoundScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int max_by_axis(cudamat* mat, cudamat* target, int axis) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != 1 || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int shared_mem_size = 32 * sizeof(float) ;
        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
        kMaxColumnwise<<<gridDim, 32, shared_mem_size>>>(mat->data_device, target->data_device, w, h);

        hipDeviceSynchronize();
    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int choose_max_and_accumulate(cudamat* mat, cudamat* acc) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !acc->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (acc->size[0] != mat->size[0] || acc->size[1] != mat->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
    kChooseMaxAndAccumulate<<<gridDim,32>>>(mat->data_device, acc->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int choose_max_by_axis(cudamat* mat, cudamat* target, int axis) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != mat->size[0] || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int shared_mem_size = 32 * sizeof(float) ;
        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
        kChooseMaxColumnwise<<<gridDim, 32, shared_mem_size>>>(mat->data_device, target->data_device, w, h);

        hipDeviceSynchronize();
    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int argmax_by_axis(cudamat* mat, cudamat* target, int axis) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != 1 || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
        kArgMaxColumnwise<<<gridDim,32>>>(mat->data_device, target->data_device, w, h);

        hipDeviceSynchronize();
    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int sqsum_by_axis(cudamat* mat, cudamat* target, int axis, float mult, float p) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != 1 || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int shared_mem_size = 32 * sizeof(float) ;
        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
        kSqSumColumnwise<<<gridDim, 32, shared_mem_size>>>(mat->data_device, target->data_device, w, h, mult, p);
        hipDeviceSynchronize();
    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int normlimit_by_axis(cudamat* mat, cudamat* target, int axis,
                                   float norm) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != mat->size[0] || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int shared_mem_size = 32 * sizeof(float) ;
        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
        kNormLimitColumnwise<<<gridDim,32, shared_mem_size>>>(mat->data_device, target->data_device, norm, w, h);
        hipDeviceSynchronize();
    } else
        return ERROR_UNSUPPORTED;
    if (checkCUDAError())
        return CUDA_ERROR;
    return 0;
}


extern int sign(cudamat* mat, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSign<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int apply_cos(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyCos<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int apply_sin(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplySin<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_sigmoid(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplySigmoid<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_tanh(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyTanh<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_abs(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyAbs<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_log_1_plus_exp(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyLog1PlusExp<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_log(cudamat* mat, cudamat* target, float tiny) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLog<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len, tiny);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_exp(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kExp<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int apply_ceil(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCeil<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int apply_floor(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kFloor<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}



extern int apply_sqrt(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSqrt<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_pow(cudamat* mat, float pow, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kPow<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_pow_matrix(cudamat* mat, cudamat* pow, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat->size[0] != pow->size[0] || mat->size[1] != pow->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kPowMatrix<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int compute_cross_entropy(cudamat* mat, cudamat* pow, cudamat* target, float tiny) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat->size[0] != pow->size[0] || mat->size[1] != pow->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCrossEntropy<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow->data_device, target->data_device, len, tiny);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int compute_cross_entropy_bernoulli(cudamat* mat, cudamat* pow, cudamat* target, float tiny) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat->size[0] != pow->size[0] || mat->size[1] != pow->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCrossEntropyBernoulli<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow->data_device, target->data_device, len, tiny);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int correct_preds(cudamat* mat, cudamat* pow, cudamat* target, float cutoff) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat->size[0] != pow->size[0] || mat->size[1] != pow->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCorrectPreds<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow->data_device, target->data_device, len, cutoff);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int reciprocal(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kReciprocal<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int dot(cudamat* mat1, cudamat* mat2, cudamat* target, float beta, float alpha) {
    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (get_leading_dimension(mat1) != get_leading_dimension(target) ||
        get_nonleading_dimension(mat2) != get_nonleading_dimension(target) ||
        get_nonleading_dimension(mat1) != get_leading_dimension(mat2)) {
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    }
    int m = get_leading_dimension(mat1),
        k = get_leading_dimension(mat2),
        n = get_nonleading_dimension(mat2);

    hipblasSgemm(get_transpose_char(mat1), get_transpose_char(mat2), 
                m, n, k,
                alpha, mat1->data_device, mat1->size[0],
                mat2->data_device, mat2->size[0],
                beta, target->data_device, target->size[0]);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    hipDeviceSynchronize();

    return 0;
}

extern int sparse_dot(cudamat_sparse* mat1, cudamat* mat2, cudamat* target, float beta, float alpha) {
    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;
    int m = mat1->size[0],
        k = mat1->size[1],
        k2 = mat2->size[0],
        n = mat2->size[1];

    if (k != k2) {
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    }
    unsigned int grid_x = m / COPY_BLOCK_SIZE;
    if (m % COPY_BLOCK_SIZE)
        grid_x++;

    unsigned int grid_y = n / COPY_BLOCK_SIZE;
    if (n % COPY_BLOCK_SIZE)
        grid_y++;

    dim3 grid(grid_x, grid_y, 1);
    dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);

    kSparseDot<<<grid, threads>>>(m, n, k, mat1->data_device.data,
        mat1->data_device.indptr,
        mat1->data_device.indices,
        mat2->data_device, target->data_device, beta, alpha);
    if (check_cublas_error())
        return CUBLAS_ERROR;

    hipDeviceSynchronize();

    return 0;
}


extern float vdot(cudamat* mat1, cudamat* mat2, int* err_code) {
    int len = mat1->size[0]*mat1->size[1];
    float res;

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans) {
        *err_code = ERROR_TRANSPOSEDNESS;
        return 0;
    }

    if (mat2->size[0] * mat2->size[1] != len) {
        *err_code = ERROR_INCOMPATIBLE_DIMENSIONS;
        return 0;
    }

    res = hipblasSdot(len, mat1->data_device, 1, mat2->data_device, 1);

    if (check_cublas_error()) {
        *err_code = CUBLAS_ERROR;
        return -1.;
    } else {
        *err_code = 0;
        return res;
    }
}

/* Perform the operation mat1 = mat1 + alpha * mat2. mat1 and mat2 must
   have the same transposedness. */
extern int add_mult(cudamat* mat1, cudamat* mat2, float alpha) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    hipblasSaxpy(len, alpha, mat2->data_device, 1, mat1->data_device, 1);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}
extern int add_mult_sign(cudamat* mat1, cudamat* mat2, float mult) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddMultSign<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, len, mult);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}


extern int add_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAdd<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int subtract_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSubtract<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int divide_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivide<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

/* Elementwise multiplication of 2 matrices */
extern int mult_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMult<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_sin_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSinDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int apply_cos_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCosDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int apply_logistic_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLogisticDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_tanh_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kTanhDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_rectified_linear_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kRectifiedLinearDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_rectified_linear_smooth_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kRectifiedLinearSmoothDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int assign_scalar(cudamat* mat, float alpha) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kAssignScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_by_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int divide_by_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivideScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int add_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern float euclid_norm(cudamat* mat, int* err_code) {
    int len = mat->size[0]*mat->size[1];

    float res =  hipblasSnrm2(len, mat->data_device, 1);

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (check_cublas_error()) {
        *err_code = CUBLAS_ERROR;
        return -1.;
    } else {
        *err_code = 0;
        return res;
    }
}
extern int selectRows(cudamat* source, cudamat* target, cudamat* indices){
    const int nRetRows = indices->size[1];

    if (nRetRows==0) return 0;

    dim3 gridDim((nRetRows+31)/32);
    dim3 blockDim(32);

    kSelectRows<<<gridDim, blockDim>>>(source->data_device, target->data_device, indices->data_device, nRetRows, source->size[0], source->size[1]);
    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}


extern int swapColumns(cudamat* source, cudamat* target, cudamat* indices1, cudamat* indices2){
    const int cols = indices1->size[1]*indices1->size[0],
                 h = source->size[0],
                 w = source->size[1];

    kSwapColumns<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(source->data_device, target->data_device, indices1->data_device, indices2->data_device, cols, w, h);
    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int setSelectedRows(cudamat* target, cudamat* source, cudamat* indices){
    const int nSetRows = indices->size[1];

    if (nSetRows==0)
        return 0;

    dim3 gridDim((nSetRows+31)/32);
    dim3 blockDim(32);

    kSetSelectedRows<<<gridDim, blockDim>>>(target->data_device, source->data_device, indices->data_device, nSetRows, target->size[0], target->size[1]);
    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int generate_translations_big_var_off(cudamat* source, cudamat* target, cudamat* off_x, cudamat* off_y, int source_w, int target_w, int num_channels) {
    dim3 kernelBlockGrid(source->size[1], 1, 1);
    dim3 kernelBlockDim(512, 1, 1);

    kGenerateTranslationsBigVarOff<<<kernelBlockGrid, kernelBlockDim>>>(source->data_device, target->data_device, off_x->data_device, off_y->data_device, source_w, target_w, num_channels);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int blockify(cudamat* source, cudamat* target, int blocksize) {
    dim3 kernelBlockGrid(source->size[1], 1, 1);
    dim3 kernelBlockDim(512, 1, 1);
    kBlockify<<<kernelBlockGrid, kernelBlockDim>>>(source->data_device, target->data_device, source->size[0], blocksize);
    if (checkCUDAError())
        return CUDA_ERROR;
    return 0;
}


extern int softmax(cudamat* mat, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    int shared_mem_size = 32 * sizeof(float) ;

    int w1 = floor(sqrt(w));
    int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
    dim3 gridDim(w1, w2, 1);
    kSoftMax<<<gridDim, 32, shared_mem_size>>>(mat->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int softmax_overwrite(cudamat* mat) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    int shared_mem_size = 32 * sizeof(float) ; 
    int w1 = floor(sqrt(w));
    int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
    dim3 gridDim(w1, w2, 1);
    kSoftMaxOverwrite<<<gridDim, 32, shared_mem_size>>>(mat->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int apply_softmax_grad(cudamat* mat, cudamat* labels, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (labels->size[0] != 1 || labels->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    kSoftMaxGrad<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, labels->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
extern int get_softmax_correct(cudamat* mat, cudamat* labels, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != 1 || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (labels->size[0] != 1 || labels->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    int w1 = floor(sqrt(w));
    int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
    dim3 gridDim(w1, w2, 1);
    kSoftMaxCorrect<<<gridDim, 32>>>(mat->data_device, labels->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int accumulate_columns(cudamat* mat, cudamat* indices, cudamat* target, float mult, int avg) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1],
                 w2 = target->size[1];

    if (!mat->on_device || !indices->on_device|| !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (indices->size[0] != 1 || indices->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (NUM_VECTOR_OP_THREADS_PER_BLOCK < w2)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    kAccumulateColumns<<<h, NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, indices->data_device, target->data_device, w, w2, h, mult, avg);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int get_softmax_cross_entropy(cudamat* mat, cudamat* labels, cudamat* target, float tiny) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != 1 || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (labels->size[0] != 1 || labels->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    kSoftMaxCrossEntropy<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, labels->data_device, target->data_device, w, h, tiny);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int expand(cudamat* source, cudamat* indices, cudamat* target){
    unsigned int h = source->size[0],
                 w = source->size[1],
                 w2 = target->size[1];

    if (!source->on_device || !indices->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (indices->size[0] != 1 || indices->size[1] != w2)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kExpand<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(source->data_device, indices->data_device, target->data_device, h, w, w2);
    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}


extern int expand_and_add(cudamat* source, cudamat* mat, cudamat* indices, cudamat* target, float mult){
    unsigned int h = source->size[0],
                 w = source->size[1],
                 w2 = mat->size[1];

    if (!source->on_device || !mat->on_device || !indices->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (indices->size[0] != 1 || indices->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (mat->size[0] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kExpandAndAdd<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(source->data_device, mat->data_device, indices->data_device, target->data_device, w, h, mult, w2);
    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}



}
