#include "hip/hip_runtime.h"
#include "cudamat_kernels.cuh"
#include "float.h"
const int NUM_THREADS = 32;

__device__ void reduceToMax(float* sdata, unsigned int tid){

  //Synchronize threads to share shared memory data
  __syncthreads();

  float mySum = sdata[tid];

  // do reduction in shared mem
  if (NUM_THREADS >= 512) { if (tid < 256) { sdata[tid] = mySum = fmaxf(mySum, sdata[tid + 256]); } __syncthreads(); }
  if (NUM_THREADS >= 256) { if (tid < 128) { sdata[tid] = mySum = fmaxf(mySum, sdata[tid + 128]); } __syncthreads(); }
  if (NUM_THREADS >= 128) { if (tid <  64) { sdata[tid] = mySum = fmaxf(mySum, sdata[tid +  64]); } __syncthreads(); }

  if (NUM_THREADS == 32){
    if (tid < 16)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (NUM_THREADS >=  32) { smem[tid] = mySum = fmaxf(mySum, smem[tid + 16]); }
      if (NUM_THREADS >=  16) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  8]); }
      if (NUM_THREADS >=   8) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  4]); }
      if (NUM_THREADS >=   4) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  2]); }
      if (NUM_THREADS >=   2) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  1]); }
    }
  }
  else
  {
    if (tid < 32)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (NUM_THREADS >=  64) { smem[tid] = mySum = fmaxf(mySum, smem[tid + 32]); }
      if (NUM_THREADS >=  32) { smem[tid] = mySum = fmaxf(mySum, smem[tid + 16]); }
      if (NUM_THREADS >=  16) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  8]); }
      if (NUM_THREADS >=   8) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  4]); }
      if (NUM_THREADS >=   4) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  2]); }
      if (NUM_THREADS >=   2) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  1]); }
    }
  }
}

__device__ void reduceToSumLocal(float* sdata, unsigned int tid)
{

  //Synchronize threads to share shared memory data
  __syncthreads();

  float mySum = sdata[tid];

  // do reduction in shared mem
  if (NUM_THREADS >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
  if (NUM_THREADS >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
  if (NUM_THREADS >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

  if (NUM_THREADS == 32){
    if (tid < 16)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (NUM_THREADS >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
      if (NUM_THREADS >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
      if (NUM_THREADS >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
      if (NUM_THREADS >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
      if (NUM_THREADS >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }
  }
  else
  {
    if (tid < 32)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (NUM_THREADS >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
      if (NUM_THREADS >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
      if (NUM_THREADS >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
      if (NUM_THREADS >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
      if (NUM_THREADS >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
      if (NUM_THREADS >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }
  }
}

/* ------------------------- Random number generation ------------------------- */

__global__ void kSeedRandom(unsigned int* rndMults, unsigned long long* rndWords, unsigned int seed) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // The initial x is the seed and the initial carry is 1
  unsigned long long rndWord = ((unsigned long long)seed << 32) + 1;
  const unsigned int rndMult = rndMults[idx];
  /*
   * Run the chain for a few steps so that all the streams have a chance
   * to differentiate. They start out generating similar random numbers
   * because all the multipliers are similar.
   */
  for(unsigned int i = 0; i < NUM_RND_BURNIN; i++) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
  }
  rndWords[idx] = rndWord;
}

__global__ void kRandomUniform(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    gData[i] = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
  }
  rndWords[idx] = rndWord;
}

__global__ void kRandomGaussian(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  float rnd1, rnd2, R, T;
  for(unsigned int i = idx; i < numElements; i += 2*NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd1 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd2 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    T = 2 * PI * rnd2;
    R = sqrtf(-2 * __logf(rnd1));
    gData[i] = R * __cosf(T);
    if (i + NUM_RND_STREAMS < numElements)
      gData[i + NUM_RND_STREAMS] = R * __sinf(T);
  }
  rndWords[idx] = rndWord;
}

__global__ void kRandomDropout(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements, float dropprob, float val) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    gData[i] = ((__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f) > dropprob ? gData[i]:val;
  }
  rndWords[idx] = rndWord;
}

__global__ void kSampleBernoulli(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    target[i] = ((__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f) < gData[i] ? 1:0;
  }
  rndWords[idx] = rndWord;
}
__global__ void kSampleBernoulliTanh(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    target[i] = ((__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f) < (1.0 + gData[i]) / 2.0 ? 1:0;
  }
  rndWords[idx] = rndWord;
}

__global__ void kSamplePoisson(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    target[i] = gData[i];
  }
  rndWords[idx] = rndWord;
}

__global__ void kSampleGaussian(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements, float mult) {

  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  float rnd1, rnd2, R, T;
  for(unsigned int i = idx; i < numElements; i += 2*NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd1 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd2 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    T = 2 * PI * rnd2;
    R = sqrtf(-2 * __logf(rnd1));
    target[i] = gData[i] + mult * R * __cosf(T);
    if (i + NUM_RND_STREAMS < numElements)
      target[i + NUM_RND_STREAMS] = gData[i + NUM_RND_STREAMS] + mult * R * __sinf(T);
  }
  rndWords[idx] = rndWord;
}

__global__ void kPerturbEnergy(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];
  float rnd;

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    target[i] = gData[i] - __logf( - __logf(rnd));
  }
  rndWords[idx] = rndWord;
}

__global__ void kPerturbProb(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];
  float rnd;

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    target[i] = - gData[i] /  __logf(rnd);
  }
  rndWords[idx] = rndWord;
}


/* ------------------------- Data copying ------------------------- */

/*
   Copy row slice from source to target. There is a block for every 32x32 chunk being copied.
 */
__global__ void kGetRowSlice(float* source, float* target, int start, int end, int width, int height) {
  const int row = start + blockIdx.x * 32 + threadIdx.x;
  const int start_col = blockIdx.y * 32;
  const int end_col = (start_col + 32 < width) ? start_col + 32: width;
  const int target_height = end - start;
  if (row < end) {
    for (int cur_col = start_col; cur_col < end_col; cur_col++)
      target[cur_col * target_height + row - start] = source[cur_col * height + row];
  }
}

__global__ void kSetRowSlice(float* source, float* target, int start, int end, int width, int height) {
  const int row = start + blockIdx.x * 32 + threadIdx.x;
  const int start_col = blockIdx.y * 32;
  const int end_col = (start_col + 32 < width) ? start_col + 32: width;
  const int source_height = end - start;
  if (row < end) {
    for (int cur_col = start_col; cur_col < end_col; cur_col++)
      target[cur_col * height + row] = source[cur_col * source_height + row - start];
    //source[cur_col * height + row - start] = target[cur_col * target_height + row];
  }
}

__global__ void kTranspose(float *odata, float *idata, int width, int height) {
  __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

  // read the matrix tile into shared memory
  unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
  unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

  if((xIndex < width) && (yIndex < height)) {
    unsigned int index_in = yIndex * width + xIndex;

    block[threadIdx.y][threadIdx.x] = idata[index_in];
  }

  __syncthreads();

  // write the transposed matrix tile to global memory
  xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
  yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

  if((xIndex < height) && (yIndex < width)) {
    unsigned int index_out = yIndex * height + xIndex;

    odata[index_out] = block[threadIdx.x][threadIdx.y];
  }
}

/* ------------------------- Mathematical operations ------------------------- */

__global__ void kLessThan(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] < mat2[i];
}

__global__ void kLessThanEq(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] <= mat2[i];
}

__global__ void kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] < val;
}

__global__ void kLessThanEqScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] <= val;
}

__global__ void kGreaterThan(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] > mat2[i];
}

__global__ void kGreaterThanEq(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] >= mat2[i];
}

__global__ void kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] > val;
}

__global__ void kGreaterThanEqScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] >= val;
}

__global__ void kUpperBound(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] > mat2[i] ? mat2[i] : mat1[i];
}

__global__ void kLowerBound(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] < mat2[i] ? mat2[i] : mat1[i];
}

__global__ void kUpperBoundScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] > val ? val:mat[i];
}

__global__ void kLowerBoundScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] < val ? val:mat[i];
}

__global__ void kSparseDot(int m, int n, int k, float *data, int* indptr, int* indices, float *dense_data, float* target, float beta, float alpha) {
  const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    const int start = indptr[row];
    const int end = indptr[row + 1];
    float sum = 0.f;
    for (int i = start; i < end; i++) {
      sum += data[i]  * dense_data[col * k + indices[i]];
    }
    const int pos = col * m + row;
    target[pos] = alpha * sum + ((beta == 0) ? 0 : beta * target[pos]);
  }
}

__global__ void kSign(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] ? copysignf(1., mat[i]) : 0;
}

__global__ void kApplySin(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = __sinf(mat[i]);
}

__global__ void kApplyCos(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = __cosf(mat[i]);
}

__global__ void kApplySigmoid(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = 1 / (1 + __expf(-mat[i]));
}

__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  float mat_i, exp2x;
  for (unsigned int i = idx; i < len; i += numThreads) {
    mat_i = mat[i];
    exp2x = __expf(2 * mat_i);
    target[i] = 1 - 2 / (exp2x + 1);
  }
}

__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] * ((mat[i] > 0) - (mat[i] < 0));
}

__global__ void kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  float mat_i;
  for (unsigned int i = idx; i < len; i += numThreads) {
    mat_i = mat[i];
    if (mat_i > 0)
      target[i] = (__logf(1 + __expf(-mat_i)) + mat_i);
    else
      target[i] = __logf(1 + __expf(mat_i));
  }
}

__global__ void kLog(float* mat, float* target, unsigned int len, float tiny) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = __logf(mat[i] + tiny);
}

__global__ void kExp(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = __expf(mat[i]);
}

__global__ void kCeil(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = ceil(mat[i]);
}

__global__ void kFloor(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = floor(mat[i]);
}

__global__ void kSqrt(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = sqrt(mat[i]);
}

__global__ void kPow(float* mat, float pow, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = powf(mat[i], pow);
}

__global__ void kPowMatrix(float* mat, float* pow, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = powf(mat[i], pow[i]);
}

__global__ void kCrossEntropy(float* mat, float* p, float* target, unsigned int len, float tiny) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = -mat[i] * __logf(p[i] + tiny);
}

__global__ void kCrossEntropyBernoulli(float* mat, float* p, float* target, unsigned int len, float tiny) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads)
    target[i] = -mat[i] * __logf(p[i] + tiny) - (1 - mat[i]) * __logf(1 - p[i] + tiny);
}

__global__ void kCorrectPreds(float* mat, float* p, float* target, unsigned int len, float cutoff) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads)
    target[i] = mat[i] * (p[i] >= cutoff) + (1 - mat[i]) * (p[i] < cutoff);
}

__global__ void kReciprocal(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = 1. / mat[i];
}

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] + vec[i % height];
  }
}
__global__ void kAddDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    tgtMat[width*i + i] = mat[width*i + i] + val;
  }
}

__global__ void kAddDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    tgtMat[width*i + i] = mat[width*i + i] + vec[i];
  }
}

__global__ void kMultDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    tgtMat[width*i + i] = mat[width*i + i] * val;
  }
}

__global__ void kMultDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    tgtMat[width*i + i] = mat[width*i + i] * vec[i];
  }
}
__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] + vec[i / height];
  }
}

__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] + mult * vec[i % height];
  }
}

__global__ void kAddRowMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] + mult * vec[i / height];
  }
}
__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] * vec[i % height];
  }
}

__global__ void kDivByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] / vec[i / height];
  }
}

__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] / vec[i % height];
  }
}

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] * vec[i / height];
  }
}
__global__ void kAddMultSign(float* a, float* b, unsigned int numEls, float mult) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    a[i] = a[i] + ((b[i] > 0) ? mult : ((b[i] < 0) ? -mult : 0));
  }
}
__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] + b[i];
  }
}

__global__ void kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] - b[i];
  }
}

__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] / b[i];
  }
}

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * b[i];
  }
}

__global__ void kCosDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = -a[i] * __sinf(b[i]);
  }
}

__global__ void kSinDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * __cosf(b[i]);
  }
}

__global__ void kLogisticDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * b[i] * (1.0 - b[i]);
  }
}

__global__ void kTanhDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * (1.0 + b[i]) * (1.0 - b[i]) * 0.5;
  }
}

__global__ void kRectifiedLinearDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * (b[i] > 0 ? 1 : 0);
  }
}

__global__ void kRectifiedLinearSmoothDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * (1 - __expf(-b[i]));
  }
}

__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) {
    dest[i] = alpha * mat[i];
  }
}

__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) {
    dest[i] = alpha;
  }
}

__global__ void kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) {
    dest[i] = mat[i] / alpha;
  }
}

__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] + alpha;
  }
}


__global__ void kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows){
  __shared__ int sourceRowIndices[32];
  const int startTargetRowI = blockIdx.x * 32;
  const int tid = threadIdx.x;
  const int localNRowIs = min(32, nRowIs-startTargetRowI);

  // cooperatively load 32 row indices
  if (tid < localNRowIs){
    sourceRowIndices[tid] = int(indices[startTargetRowI + tid]);
    if (sourceRowIndices[tid]<0)
      sourceRowIndices[tid] += nSourceRows;
    if (sourceRowIndices[tid]<0 || sourceRowIndices[tid]>=nSourceRows)
      sourceRowIndices[tid] = -1;
  }
  __syncthreads();

  // copy 32 rows
  for (int i=0; i<localNRowIs; i++){
    const int targetRowI = startTargetRowI + i, sourceRowI = sourceRowIndices[i];
    for (int colI=tid; colI<nCols; colI+=32)
      target[targetRowI * nCols + colI] = sourceRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
  }
}

__global__ void kSwapColumns(float* source, float* target, float* indices1, float* indices2, int cols, int width, int height){
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  float temp;
  unsigned int column, row, source_pos, target_pos;
  for (unsigned int i = idx; i < height * cols; i += numThreads) {
    column = i / height;
    row = i % height;
    source_pos = height * (int)indices1[column] + row;
    target_pos = height * (int)indices2[column] + row;
    temp = source[source_pos];
    source[source_pos] = target[target_pos];
    target[target_pos] = temp;
  }
}

__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows){
  __shared__ int targetRowIndices[32];
  const int startSourceRowI = blockIdx.x * 32;
  const int tid = threadIdx.x;
  const int localNRowIs = min(32, nRowIs-startSourceRowI);

  // cooperatively load 32 row indices
  if (tid < localNRowIs){
    targetRowIndices[tid] = int(indices[startSourceRowI + tid]);
    if (targetRowIndices[tid]<0)
      targetRowIndices[tid] += nTargetRows;
    if (targetRowIndices[tid]<0 || targetRowIndices[tid]>=nTargetRows)
      targetRowIndices[tid] = -1;
  }
  __syncthreads();

  // copy 32 rows
  for (int i=0; i<localNRowIs; i++){
    const int sourceRowI = startSourceRowI + i, targetRowI = targetRowIndices[i];
    for (int colI=tid; colI<nCols; colI+=32)
      target[targetRowI * nCols + colI] = targetRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
  }
}

__global__ void kBlockify(float* source, float* target, int numdims, int blocksize) {
  const unsigned int idx = threadIdx.x;
  const unsigned int numThreads = blockDim.x;
  const int off = blockIdx.x * numdims;

  for (unsigned int target_ind = idx; target_ind < numdims; target_ind += numThreads) {
    const int block = target_ind / blocksize;
    target[off + target_ind] = source[off + block * blocksize];
  }
}

__global__ void kGenerateTranslationsBigVarOff(float* source, float* target, float* off_x_arr, float* off_y_arr, int source_w, int target_w, int num_channels) {
  const unsigned int idx = threadIdx.x;
  const unsigned int numThreads = blockDim.x;

  int target_x, target_y;
  int pad = (source_w - target_w)/2;
  int target_tile_size = target_w * target_w;
  int source_tile_size = source_w * source_w;

  int off_x = off_x_arr[blockIdx.x];
  int off_y = off_y_arr[blockIdx.x];
  int target_off = blockIdx.x * target_tile_size;
  int source_off = blockIdx.x * source_tile_size + (pad + off_x) * source_w + (pad + off_y);

  for (unsigned int target_ind = idx; target_ind < target_tile_size; target_ind += numThreads) {
    target_x = target_ind / target_w;
    target_y = target_ind - target_x * target_w;

    for (unsigned int ch = 0; ch < num_channels; ch += 1) {
      target[num_channels*(target_off + target_x * target_w + target_y) + ch] = source[num_channels*(source_off + target_x * source_w + target_y) + ch];
    }
  }
}

__global__ void kSoftMaxGrad(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width*height; i += numThreads) {
    target[i] = mat[i] - ((int)labels[i / height] == i % height ? 1 : 0);
  }
}

__global__ void kSoftMaxCrossEntropy(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    target[i] = -__logf(mat[height * i + (int)labels[i]] + tiny);
  }
}

__global__ void kSoftMaxCorrect(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target[column] = (cur_argmax == (int)labels[column]) ? 1 : 0;
    }
  }
}


__global__ void kSoftMax(float* mat, float* target, unsigned int width, unsigned int height) {
  extern __shared__ float max_vals[] ;
  float cur_max = -FLT_MAX;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    max_vals[threadIdx.x]=-FLT_MAX;
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    reduceToMax(max_vals, threadIdx.x);
    __syncthreads();
    cur_max = max_vals[0] ; 
    __syncthreads();
    val = 0;
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val += __expf(cur_data[i]-cur_max);
    }
    max_vals[threadIdx.x] = val;
    reduceToSumLocal(max_vals, threadIdx.x);
    __syncthreads();
    float norm = max_vals[0] ; 
    float *cur_target = &target[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_target[i] = __expf(cur_data[i]-cur_max) / norm ;
    }
  }
}

__global__ void kSoftMaxOverwrite(float* mat, unsigned int width, unsigned int height) {
  extern __shared__ float max_vals[] ;
  float cur_max = -FLT_MAX;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    max_vals[threadIdx.x]=-FLT_MAX;
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    reduceToMax(max_vals, threadIdx.x);
    __syncthreads();
    cur_max = max_vals[0] ;
    __syncthreads();
    val = 0;
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_data[i] = __expf(cur_data[i]-cur_max);
      val += cur_data[i];
    }
    max_vals[threadIdx.x] = val;
    reduceToSumLocal(max_vals, threadIdx.x);
    __syncthreads();
    float norm = max_vals[0] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_data[i] /= norm;
    }
  }
}

__global__ void kChooseMaxAndAccumulate(float* mat, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    float *target_data = &target[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target_data[cur_argmax] += 1;
    }
  }
}
__global__ void kChooseMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    float *target_data = &target[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      target[i] = 0;
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target_data[cur_argmax] = 1;
    }
  }
}

__global__ void kMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
  extern __shared__ float max_vals[] ;
  float cur_max = -FLT_MAX;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) cur_max = val;
    }
    max_vals[threadIdx.x] = cur_max;
    reduceToMax(max_vals, threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0) target[column] = max_vals[0];
  }
}

__global__ void kArgMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target[column] = cur_argmax;
    }
  }
}

__global__ void kSqSumColumnwise(float* mat, float* target, unsigned int width, unsigned int height, float mult, float p) {
  extern __shared__ float sum_vals[];
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float cur_sum = 0;
    float *cur_data = &mat[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_sum += cur_data[i]*cur_data[i];
    }
    sum_vals[threadIdx.x] = cur_sum;
    reduceToSumLocal(sum_vals, threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0) target[column] = p * target[column] + mult * sum_vals[0];
  }
}

__global__ void kNormLimitColumnwise(float* mat, float* target, float norm, unsigned int width, unsigned int height) {
  extern __shared__ float sum_vals[];
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float cur_sum = 0;
    float *cur_data = &mat[column * height] ; 
    float *target_data = &target[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_sum += cur_data[i]*cur_data[i];
    }
    sum_vals[threadIdx.x] = cur_sum;
    reduceToSumLocal(sum_vals, threadIdx.x);
    __syncthreads();
    cur_sum = sqrt(sum_vals[0]);
    cur_sum = (cur_sum < norm) ? 1: (norm / cur_sum);
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      target_data[i] = cur_data[i] * cur_sum;
    }
    __syncthreads();
  }
}

__global__ void kExpand(float* source, float* indices, float* target, int height, int width, int target_width){
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < target_width*height; i += numThreads) {
    const int pos = height * (int)indices[i / height] + i % height;
    target[i] = (pos < height * width)? source[pos] : 1.0/0.0 - 1.0/0.0;
  }
}


__global__ void kExpandAndAdd(float* source, float* mat, float* indices, float* target, int width, int height, float mult, int width2){
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width*height; i += numThreads) {
    const int pos = height * (int)indices[i / height] + i % height;
    target[i] = (pos < height * width2)? source[i] + mult * mat[pos] : 1.0/0.0 - 1.0/0.0;
  }
}

__global__ void kAccumulateColumns(float* mat, float* indices, float* target, int mat_width, int target_width, int height, float mult, int avg){
  const int row = gridDim.x * blockIdx.y + blockIdx.x;
  const int column = threadIdx.x;
  if (row < height && column < target_width) {
    float cur_sum = 0.0;
    unsigned int count = 0;
    for (unsigned int i = 0; i < mat_width; i ++) {
      count += ((int)indices[i] == column) ? 1 : 0 ;
      cur_sum += ((int)indices[i] == column) ? mat[row + i * height] : 0 ;
    }
    target[row + height * column] = mult * cur_sum / ((avg == 1 && count > 0) ? count : 1);
  }
}
